#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;

__global__ void mandelKernel(int* result, float lowerX, float lowerY, float stepX, float stepY, int iter, int resX, int resY, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    
    int thisY = threadIdx.y + blockIdx.y * blockDim.y;
    int thisX = threadIdx.x + blockIdx.x * blockDim.x;
    // int* row = (int*)((char*)result+thisX*pitch);
    if (thisY >= resY || thisX >= resX)
        return;
    float x = lowerX + (float)thisX * stepX;
    float y = lowerY + (float)thisY * stepY;
    float x0 = x, y0 = y;
    
    int i;
    for (i = 0; i < iter; i++)
    {
        if (x * x + y * y > 4.f)
            break;
        
        float new_x = x * x - y * y;
        float new_y = 2.f * x * y;
        x = new_x + x0;
        y = new_y + y0;
    }
    result[thisY*pitch+thisX] = i;
    return;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
		float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int N = resX*resY;
		// �ӽ�host�O����
	  size_t in = N * sizeof(int);
    int* result;
    
    hipHostAlloc(&result, in, hipHostMallocPortable);
    // �ӽ�device�O����
		int *d_result;
		size_t pitch;
    // hipMalloc(&d_result, in);
    hipMallocPitch((void **)&d_result, &pitch, sizeof(int)*resX, (size_t) resY);
    
    // �w�qkernel������t�m
    dim3 blockSize(32, 32);
    dim3 gridSize((pitch/sizeof(float)+blockSize.x-1)/blockSize.x,(resY+blockSize.y-1)/blockSize.y);
    // ����kernel
    mandelKernel << < gridSize, blockSize >> >(d_result, lowerX, lowerY, stepX, stepY, maxIterations, resX, resY, pitch/sizeof(float));
    
    // �Ndevice�o�쪺���G������host
    hipMemcpy2D(result, sizeof(float)*resX,d_result, pitch, sizeof(float)*resX, resY, hipMemcpyDeviceToHost);

    memcpy(img, result, in);
    
    // ����device�O����
    hipFree(d_result);
    // ����host�O����
    hipHostFree(result);
}
