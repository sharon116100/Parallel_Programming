#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;

__global__ void mandelKernel(int* result, float lowerX, float lowerY, float stepX, float stepY, int iter, int resX, int resY) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int thisY = threadIdx.y + blockIdx.y * blockDim.y;
    int thisX = threadIdx.x + blockIdx.x * blockDim.x;
    int index = resX * thisY + thisX;
    if (thisY >= resY || thisX >= resX)
        return;
    float x = lowerX + (float)thisX * stepX;
    float y = lowerY + (float)thisY * stepY;
    float x0 = x, y0 = y;
    
    int i;
    for (i = 0; i < iter; i++)
    {
        if (x * x + y * y > 4.f)
            break;
        
        float new_x = x * x - y * y;
        float new_y = 2.f * x * y;
        x = new_x + x0;
        y = new_y + y0;
    }
    result[index] = i;
    return;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int N = resX*resY;
		// �ӽ�host�O����
	  size_t in = N * sizeof(int);
    int* result = (int*)malloc(in);
    
    // �ӽ�device�O����
		int *d_result;
    hipMalloc(&d_result, in);
    
    // �w�qkernel������t�m
    dim3 blockSize(16, 16);
    dim3 gridSize((resX + blockSize.x - 1) / blockSize.x, (resY + blockSize.y - 1) / blockSize.y);
    // ����kernel
    mandelKernel << < gridSize, blockSize >> >(d_result, lowerX, lowerY, stepX, stepY, maxIterations, resX, resY);
    
    // �Ndevice�o�쪺���G������host
    hipMemcpy(result, d_result, in, hipMemcpyDeviceToHost);
    memcpy(img, result, in);
    
    // ����device�O����
    hipFree(d_result);
    // ����host�O����
    free(result);
}
